#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <tuple>
#include <iostream>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __shared__
#endif

using namespace std;

__device__ float distance(float *testStart, float *testEnd, float *trainStart, float *trainEnd) {
//     float sum = 0;

//     for (int i = 0; i < a->size() - 1; i++) {
//         float diff = (a->get(i)->operator float() - b->get(i)->operator float());
//         sum += diff * diff;
//     }

//     return sum;
}

__global__ void KNN(float *train, float *test, int *predictions, int k, int numAttr, int numClasses, int *datasetSizes) {
    // Implements a sequential kNN where for each candidate query an in-place priority queue is maintained to identify the kNN's.

    // stores k-NN candidates for a query vector as a sorted 2d array. First element is inner product, second is class.
    float *candidates = (float *) calloc(k * 2, sizeof(float));
    for (int i = 0; i < 2 * k; i++) {
        candidates[i] = FLT_MAX;
    }

    // Stores bincounts of each class over the final set of candidate NN
    int *classCounts = (int *) calloc(numClasses, sizeof(int));

    // printf(*test + 0, *test + 0 + 8, test[0], test[8]);

    for (int queryIndex = 0; queryIndex < datasetSizes[1]; queryIndex++) {
        for (int keyIndex = 0; keyIndex < datasetSizes[0]; keyIndex++) {

            // float dist = distance(test->get_instance(queryIndex), train->get_instance(keyIndex));
            float dist = distance(test + queryIndex, test + queryIndex + numAttr, train + keyIndex, train + keyIndex + numAttr);

            // Add to our candidates
            for (int c = 0; c < k; c++) {
                if (dist < candidates[2 * c]) {
                    // Found a new candidate
                    // Shift previous candidates down by one
                    for (int x = k - 2; x >= c; x--) {
                        candidates[2 * x + 2] = candidates[2 * x];
                        candidates[2 * x + 3] = candidates[2 * x + 1];
                    }

                    // Set key vector as potential k NN
                    candidates[2 * c] = dist;
                    candidates[2 * c + 1] = train->get_instance(keyIndex)->get(train->num_attributes() - 1)->operator float(); // class value

                    break;
                }
            }
        }

        // Bincount the candidate labels and pick the most common
        for (int i = 0; i < k; i++) {
            classCounts[(int)candidates[2 * i + 1]] += 1;
        }

        int max = -1;
        int max_index = 0;
        for (int i = 0; i < num_classes; i++) {
            if (classCounts[i] > max) {
                max = classCounts[i];
                max_index = i;
            }
        }

        predictions[queryIndex] = max_index;

        for (int i = 0; i < 2 * k; i++) {
            candidates[i] = FLT_MAX;
        }
        memset(classCounts, 0, num_classes * sizeof(int));
    }
}

int *computeConfusionMatrix(int *predictions, ArffData *dataset) {
    int *confusionMatrix = (int *) calloc(dataset->num_classes() * dataset->num_classes(), sizeof(int)); // matrix size numberClasses x numberClasses

    for (int i = 0; i < dataset->num_instances(); i++) // for each instance compare the true class and predicted class {
        int trueClass = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
        int predictedClass = predictions[i];

        confusionMatrix[trueClass * dataset->num_classes() + predictedClass]++;
    }

    return confusionMatrix;
}

float computeAccuracy(int *confusionMatrix, ArffData *dataset) {
    int successfulPredictions = 0;

    for (int i = 0; i < dataset->num_classes(); i++) {
        successfulPredictions += confusionMatrix[i * dataset->num_classes() + i]; // elements in the diagonal are correct predictions
    }

    return successfulPredictions / (float)dataset->num_instances();
}

int main(int argc, char *argv[]) {
    int k = strtol(argv[3], NULL, 10);

    // Open the datasets
    ArffParser parserTrain(argv[1]);
    ArffParser parserTest(argv[2]);
    ArffData *train = parserTrain.parse();
    ArffData *test = parserTest.parse();

    int numAttr = train->num_attributes();
    int numClasses = train->num_classes();
    int datasetSizes[2] = {(int)train->num_instances(), (int)test->num_instances()};

    // Allocate host memory
    // float (*h_train_instances)[numAttr] = (float * [numAttr]) malloc(sizeof(float[datasetSizes[0]][numAttr]));
    float *h_train_instances = (float *) malloc(datasetSizes[0] * numAttr * sizeof(float));
    // float (*h_test_instances)[numAttr] = (float * [numAttr]) malloc(sizeof(float[datasetSizes[1]][numAttr]));
    float *h_test_instances = (float *) malloc(datasetSizes[1] * numAttr * sizeof(float));
    int *h_predictions = (int *) malloc(datasetSizes[1] * sizeof(int));

    for (int i = 0; i < datasetSizes[0]; i++) {
        for (int j = 0; j < numAttr; j++)
        {
            h_train_instances[i * numAttr + j] = train->get_instance(i)->get(j)->operator float();

            if (i < datasetSizes[1])
                h_test_instances[i * numAttr + j] = test->get_instance(i)->get(j)->operator float();
        }
    }

    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Allocate device memory
    float *d_train_instances, *d_test_instances;
    int *d_predictions;

    hipMalloc(&d_train_instances, datasetSizes[0] * numAttr * sizeof(float));
    hipMalloc(&d_test_instances, datasetSizes[1] * numAttr * sizeof(float));
    hipMalloc(&d_predictions, datasetSizes[1] * sizeof(int));

    // Copy host memory to device memory
    hipMemcpy(d_train_instances, h_train_instances, datasetSizes[0] * numAttr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_test_instances, h_test_instances, datasetSizes[1] * numAttr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_predictions, h_predictions, datasetSizes[1] * sizeof(int), hipMemcpyHostToDevice);

    // Configure the blocks and grid sizes
    int threadsPerBlock = 64;
    int gridSize = (datasetSizes[1] + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel function
    KNN<<<gridSize, threadsPerBlock>>>(d_train_instances, d_test_instances, d_predictions, k, numAttr, numClasses, datasetSizes);

    // Transfer device results to host memory
    hipMemcpy(h_predictions, d_predictions, datasetSizes[1] * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    // Compute the confusion matrix
    // int* confusionMatrix = computeConfusionMatrix(predictions, test);
    // Calculate the accuracy
    // float accuracy = computeAccuracy(confusionMatrix, test);

    // printf("The %i-NN classifier for %lu test instances on %lu train instances required %f ms CPU time. Accuracy was %.4f\n", k, test->num_instances(), train->num_instances(), milliseconds, accuracy);

    // printf("%d, %d, %d, %d", k, datasetSizes[1], threadsPerBlock, gridSize);

    // Free memory
    hipFree(d_train_instances);
    hipFree(d_test_instances);
    hipFree(d_predictions);
    free(h_train_instances);
    free(h_test_instances);
    free(h_predictions);

    return 0;
}
